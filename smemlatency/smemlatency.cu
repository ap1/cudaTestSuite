#include "hip/hip_runtime.h"

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <cutil.h>
#include <ctime>
#include <assert.h>

#define ALLOCATED_SMEM_INDICES  (1024)

// --- kernel --- //

__global__ void testSMemLatency(
        float* random_output_array, long N_SMEM_FETCHES){

    // idea is simple: thread 0 loads one value from smem
    // in a dependent fashion

    __shared__ float sharedVals[ALLOCATED_SMEM_INDICES];

    // create a circular linked list
    // (use all threads)
    int idx = threadIdx.x;
    while(idx < ALLOCATED_SMEM_INDICES){
        sharedVals[idx] = (float)((idx+1)%ALLOCATED_SMEM_INDICES);
        idx += blockDim.x;
    }
    __syncthreads();

    float sum=1.0f;

    // to avoid bank conflicts,
    // only use one thread
    if(threadIdx.x==0){
#pragma unroll 100
        for(long i=0; i<N_SMEM_FETCHES; i++){
            sum = sharedVals[(int)sum];
        }
    }

    __syncthreads();

    if(threadIdx.x==0){
        random_output_array[blockIdx.x] = sum;
    }
}


// --- wrappers --- //

clock_t startTime;

void    startTimer();
float   getTimer();
void    runSMemLatencyTest();


int main(){
    hipSetDevice(0);
    runSMemLatencyTest();
}

void runSMemLatencyTest(){
    int nBlocks, nThreads;

    nBlocks  = 1;
    nThreads = 32;

    // allocate temp output array
    float* random_output_array;
    hipMalloc((void**)&random_output_array, nBlocks*sizeof(float));

    // trial run
    testSMemLatency<<<nBlocks, nThreads>>>(random_output_array,100);
    hipDeviceSynchronize(); CUT_CHECK_ERROR("");


    for(int run=0; run<10; run++){

        long baseSmemFetchCount = 500000;
        long sMemFetchCount = baseSmemFetchCount * (run+1);
        int nTests = 100;

        assert(sMemFetchCount % 100 == 0);

        startTimer();
        for(int i=0; i<nTests; i++){
            testSMemLatency<<<nBlocks, nThreads>>>
                (random_output_array, sMemFetchCount);
        }
        hipDeviceSynchronize();
        float timer = getTimer();
        CUT_CHECK_ERROR("");
        printf("%d, %f, // %d tries, total %f ms, %0.2fms per kernel\n", 
                sMemFetchCount, timer/(float)nTests,
                nTests, timer, timer/(float)nTests);
    }

}


// reset clock
void startTimer(){
    startTime = clock();
}

// result is in ms
float getTimer(){
    return (1000.0f*((float)(clock()-startTime))/(float)CLOCKS_PER_SEC);
}
